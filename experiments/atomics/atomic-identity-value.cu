#include "hip/hip_runtime.h"
#include "mincu.hpp"

#include <array>
#include <bit>
#include <tuple>

using namespace mincu;

static const unsigned ITRS = 16;

extern "C"
__global__ void count_oups_hw(
    uint32_t *oups, const uint32_t *inps, uint32_t zero)
{
  const size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t inp = inps[gid];
  // hammer the atomic ITRS times with the same value
  for (unsigned i = 0; i < ITRS; i++) {
    auto inp_i = inp + i * zero;
    atomicAdd(&oups[gid + i * zero], inp_i);
  }
}

extern "C"
__global__ void count_oups_sw(
    uint32_t *oups, const uint32_t *inps, uint32_t zero)
{
  const size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t inp = inps[gid];
  // hammer the atomic ITRS times with the same value
  for (unsigned i = 0; i < ITRS; i++) {
    auto inp_i = inp + i * zero;
    if (inp_i != 0) {
      atomicAdd(&oups[gid + i * zero], inp_i);
    }
  }
}


struct opts {
  int verbosity = 0;
  int iterations = 1;
  bool check = false;
  size_t blocks_per_grid = 1024;
  size_t threads_per_block = 256;

  bool normal() const {return verbosity >= 0;}
  bool verbose() const {return verbosity >= 1;}
  bool debug() const {return verbosity >= 2;}
  bool verbose_debug() const {return verbosity >= 3;}
};

constexpr opts DFT_OPTS;


static void referee_oups(
  const opts &os, uint32_t *oups, const uint32_t *inps, unsigned delta)
{
  size_t total_threads = os.blocks_per_grid * os.threads_per_block;

  int errs = 0;
  for (size_t gid = 0; gid < total_threads; gid++) {
    auto ref_oup = inps[gid] * ITRS;
    if (oups[gid] != ref_oup) {
      std::cerr << format("oups[", gid, "]: has # ",
          oups[gid], ", but we expected ", ref_oup, "\n");
      errs++;
    }
  }
  if (errs)
    fatal("mismatched");
}


using dispatch_wrapper =
    std::function<void (size_t,size_t,uint32_t *,const uint32_t *)>;


static void run_test(
    const opts &os,
    std::string tnm,
    std::function<unsigned(size_t)> init,
    dispatch_wrapper dispatch)
{
  if (os.debug()) {
    std::cout << "============= " << tnm << " starting\n";
  }

  umem<uint32_t> oups {os.blocks_per_grid * os.threads_per_block};
  const umem<uint32_t> inps {os.blocks_per_grid * os.threads_per_block, init};

  const double mb = (double)(inps.size() * sizeof(inps[0])) / 1024.0 / 1024.0;

  float min_t = 0.0f;
  for (int i = 0; i < os.iterations; i++) {
    oups.init(const_seq(0u));
    float t =
        time_dispatch_s(
          [&] {
            dispatch(os.blocks_per_grid, os.threads_per_block, oups, inps);
          });
    min_t = i == 0 ? t : std::min(min_t, t);
    if (os.debug()) {
      std::cout << "run[" << i << "]: " << frac(mb / min_t, 3) << "  MB/s\n";
    }
  }

  const double min_mb_s = mb / min_t;
  std::cout << coll<std::string>(tnm + ": ", 12) <<
      frac(min_mb_s, 3) << "  MB/s\n";

  if (os.verbose()) {
    std::cout << "  elems:  " << inps.size() << " elems\n";
    std::cout << "  mem:    " << frac(mb, 3) << " MB\n";
    std::cout << "  time:   " << frac(min_t, 5) << " s\n";
  }

  if (os.verbose_debug()) {
    std::cout << "INPS:\n";
    inps.str(std::cout, 8);
  }
  if (os.verbose_debug()) {
    std::cout << "OUPS:\n";
    oups.str(std::cout, 8);
  }

  if (os.check) {
    referee_oups(os, oups, inps, 0);
    if (os.normal()) {
      std::cout << "referee check passed\n";
    }
  }
}

using test = std::tuple<std::string,std::function<unsigned(size_t)>,dispatch_wrapper>;
static const test ALL_TESTS[] {
  {"hw-00-zeros",
      [] (size_t ix) {return 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_hw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"hw-01-zeros",
      [] (size_t ix) {return ix % 32 == 0 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_hw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"hw-08-zeros",
      [] (size_t ix) {return ix % 32 < 8 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_hw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"hw-16-zeros",
      [] (size_t ix) {return ix % 32 < 16 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_hw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"hw-24-zeros",
      [] (size_t ix) {return ix % 32 < 24 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_hw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"hw-31-zeros",
      [] (size_t ix) {return ix % 32 < 31 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_hw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"hw-32-zeros",
      [] (size_t ix) {return 0;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_hw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"sw-00-zeros",
      [] (size_t ix) {return 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_sw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"sw-01-zeros",
      [] (size_t ix) {return ix % 32 == 0 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_sw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"sw-08-zeros",
      [] (size_t ix) {return ix % 32 < 8 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_sw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"sw-16-zeros",
      [] (size_t ix) {return ix % 32 < 16 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_sw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"sw-24-zeros",
      [] (size_t ix) {return ix % 32 < 24 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_sw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"sw-31-zeros",
      [] (size_t ix) {return ix % 32 < 31 ? 0 : 1;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_sw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
  {"sw-32-zeros",
      [] (size_t ix) {return 0;},
      [] (size_t blocks_per_grid, size_t block_size, uint32_t *oups, const uint32_t *inps) {
        count_oups_sw<<<blocks_per_grid,block_size>>>(oups, inps, 0);
      }},
};

int main(int argc, const char* argv[])
{
  struct opts os;
  std::vector<test> tests;

  for (int i = 1; i < argc; i++) {
    std::string arg = argv[i];
    std::string key = arg, val;
    auto eq = arg.find('=');
    if (eq != std::string::npos) {
      key = arg.substr(0, eq + 1); // include the =
      val = arg.substr(eq + 1);
    }
    auto bad_opt = [&](const char *msg) {
      fatal(arg, ": ", msg);
    };

    if (arg == "-h" || arg == "--help") {
      std::stringstream tss;
      tss << "all";
      for (const auto &[tnm,_,__] : ALL_TESTS) {
        tss << " | " << tnm;
      }
      std::stringstream uss;
      uss <<
        "usage: atomic-identity-value.exe [OPTS] TESTS+\n"
        "where OPTS:\n"
        "  -bpg=INT              blocks per grid (defaults to " << DFT_OPTS.blocks_per_grid << ")\n"
        "  -tpb=INT              threads per blocks (defaults to " << DFT_OPTS.threads_per_block << ")\n"
        "  --check               referee the output on CPU\n"
        "  -i/--iterations=INT   number of runs to take the min of\n"
        "  -v/-v2/-v3            verbosity/debug\n"
        "and TESTS are: " << tss.str() << "\n"
        "EXAMPLES:\n"
        " % ... -bpg=1k all\n"
        "  generates and processes 1k blocks per grid on all tests\n"
        "";
      std::cout << uss.str();
      return EXIT_SUCCESS;
    } else if (arg == "--check") {
      os.check = true;
    } else if (key == "-bpg=") {
      os.blocks_per_grid = parse_integral_positive<size_t>(val, true);
    } else if (key == "-tpb=") {
      os.threads_per_block = parse_integral_positive<size_t>(val, true);
    } else if (key == "-i=" || key == "--iterations=") {
      os.iterations = parse_integral_positive<int>(val, false);
    } else if (arg == "-v") {
      os.verbosity = 1;
    } else if (arg == "-v2") {
      os.verbosity = 2;
    } else if (arg == "-v3") {
      os.verbosity = 3;
    } else if (arg == "all") {
      for (const auto &t : ALL_TESTS) {
        tests.push_back(t);
      }
    } else {
      bool found = false;
      for (const auto &[tnm,init,disp] : ALL_TESTS) {
        if (tnm == arg) {
          tests.emplace_back(tnm, init, disp);
          found = true;
          break;
        }
      }
      if (!found) {
        bad_opt("invalid argument (try -h)");
      }
    }
  } // for

  if (tests.empty()) {
    fatal("expected test name (try -h)");
  }

  for (const auto &[tnm,init,disp] : tests) {
    run_test(os, tnm, init, disp);
  }

  return EXIT_SUCCESS;
}
