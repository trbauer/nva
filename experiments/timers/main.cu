#include "hip/hip_runtime.h"
#include "mincu.hpp"
#include "timers.cuh"

#include <hip/hip_runtime_api.h>

#include <algorithm>
#include <array>
#include <cctype>
#include <cstdint>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <map>
#include <ostream>
#include <sstream>
#include <string>
#include <thread>
#include <vector>


using namespace mincu;

/*
static std::string fmtSmid(smid_t smid) {
  return "@" + std::to_string(smid);
}
template <typename T>
static std::string fmtKey(T t) {
    std::stringstream ss;
    ss << t << ":";
    return ss.str();
}
*/



struct statistics {
  int64_t n = 0;
  int64_t sm = 0;
  double av = 0.0;
  double md = 0.0;
  double mn = 0.0, mx = 0.0;
  double va = 0.0;

  template <typename T>
  static statistics construct(const umem<T> &oup)
  {
    return construct<T>(oup, oup.size());
  }
  template <typename T>
  static statistics construct(const T *oup, size_t _n)
  {
    statistics s;
    s.add_all(oup, _n);
    return s;
  }

  template <typename T>
  void add_all(const T *oup, size_t _n) {
    n = _n;
    if (n == 0)
      return;

    sm = 0;
    T _mx = oup[0], _mn = oup[0];
    std::vector<T> ord(n);
    for (size_t i = 0; i < n; i++) {
      auto e = oup[i];
      sm += e;
      ord.push_back(e);
      _mn = std::min<T>(_mn, e);
      _mx = std::max<T>(_mx, e);
    }
    mn = (double)_mn;
    mx = (double)_mx;
    av = (double)sm/n;

    std::sort(ord.begin(), ord.end());
    if (n % 2) {
      md = ord[n/2];
    } else {
      md = (ord[n/2 - 1] + ord[n/2])/2.0;
    }

    int64_t dvsm = 0;
    for (size_t i = 0; i < n; i++) {
      auto e = oup[i];
      dvsm += (e - av)*(e - av);
    }
    va = (double)dvsm/n;
  }

  /////////////////////////////////////
  // average
  double avg() const {return av;}
  // sum
  int64_t sum() const {return sm;}

  /////////////////////////////////////
  // ordering
  //
  // minimum
  double min() const {return mn;}
  // median
  double med() const {return md;}
  // maximum
  double max() const {return mx;}

  /////////////////////////////////////
  // spread
  // variance
  double var() const {return va;}
  // standard deviation
  double sdv() const {return sqrt(var());}
  // standard error of the mean
  double sem() const {return sdv()/sqrt((double)n);}
  // realtive standard error
  double rse() const {return sem()/avg();}
};


// %globaltimer
//   https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#special-registers-globaltimer

// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#special-registers-warpid
//   %warpid  %nwarpid

// %smid and %nsmid  (may not be contiguous)
// %gridid

// __nanosleep
// The sleep duration is approximated, but guaranteed to be in the
// interval [0, 2*t].  The implementation may reduce the sleep duration for
// individual threads within a warp such that all sleeping threads in the
// warp wake up together.


using block_dist_map_t = std::map<smid_t,std::vector<std::pair<block_id_t,ticks_t>>>;



// dispatched block id with value
struct dbiv {
  dbi       id;
  ticks_t   value;

  dbiv(block_id_t bid, smid_t smid, ticks_t v) : id(bid, smid), value(v) { }
  dbiv(uint32_t idbits, ticks_t v) : id(idbits), value(v) { }
};

struct rec {
  static const int BLOCK_SIZE_COLUMN = 8;
  static const int DATA_COLUMN = 12;

  int                   blocks;
  std::vector<dbiv>     elems;

  rec(int _blocks,
      const umem<uint32_t> &dbids_buf,
      const umem<ticks_t> *times_buf = nullptr)
      : blocks(_blocks)
  {
    elems.reserve(blocks);
    for (size_t i = 0; i < blocks; i++) {
      auto idbits = dbids_buf[i];
      if (times_buf) {
        auto t = (*times_buf)[i];
        elems.emplace_back(idbits, t);
      } else {
        elems.emplace_back(idbits, 0);
      }
    }
  }
  rec(int _blocks,
      const umem<uint32_t> &ids_buf,
      const umem<ticks_t> &times_buf) : rec(_blocks, ids_buf, &times_buf) { }

  block_dist_map_t block_distribution() const {
    block_dist_map_t m;
    for (dbiv d : elems) {
      // const auto &elems = m[d.id.sm];
      // std::vector<std::pair<block_id_t,ticks_t>> &es = m[d.id.sm];
      block_dist_map_t::mapped_type &es = m[d.id.sm];
      // auto &es = m[d.id.sm];
      es.push_back(std::pair<block_id_t,ticks_t>(d.id.block, d.value));
    }
    return m;
  }
  void format_block_distribution(std::ostream &os) const {
    const auto m = block_distribution();
    // for (const std::pair<smid_t,std::vector<std::pair<block_id_t,ticks_t>>> &bin : m) {
    for (const auto &bin : m) {
      auto smid = bin.first;
      const block_dist_map_t::mapped_type &elems = bin.second;
      os << std::setw(BLOCK_SIZE_COLUMN) << "@" + std::to_string(smid) + ":";
      for (const auto &val : elems) {
        os << "  " << std::setw(5) << std::right << "#" + std::to_string(val.first);
        // os << " = " << std::setw(12) << std::right << val.second;
      }
      os << "\n";
    }
  }

  void format_block_distribution_json(std::ostream &os) const {
    bool first = true;
    os << "[";

    const auto m = block_distribution();
    for (const auto &bin : m) {
      auto smid = bin.first;
      const block_dist_map_t::mapped_type &elems = bin.second;
      for (const auto &val : elems) {
        if (first) {
          first = false;
        } else {
          os << "\n,";
        }
        auto block_id = val.first;
        auto ts = val.second;
        os << "{\"pid\":" << smid << ", \"ph\":\"I\", \"name\":\"#" << block_id << "\", \"ts\":" << ts << "}";
      }
    }

    os << "]";
  }

  void str(std::ostream &os) const {
    if (!elems.empty()) {
      os << std::setw(BLOCK_SIZE_COLUMN) << (std::to_string(blocks) + ":");
      for (size_t i = 0; i < elems.size(); i++) {
        os << "  " << std::setw(DATA_COLUMN) << elems[i].value;
      }
      os << "\n";
    }

    os << std::setw(BLOCK_SIZE_COLUMN) << (std::to_string(blocks) + ":");
    for (size_t i = 0; i < elems.size(); i++) {
      std::stringstream ss;
      ss << elems[i].id.block << "@" << elems[i].id.sm;
      os << "  " << std::setw(DATA_COLUMN) << ss.str();
    }
    os << "\n";
  }
};



extern "C" __global__ void glob_get_nsmid(int *nsmid);

static int query_nsmid()
{
  umem<int> nsmid_buf(1);
  glob_get_nsmid<<<1,1>>>(nsmid_buf);
  return nsmid_buf[0];
}



extern "C" __global__ void glob_block_dist(int64_t *times, uint32_t *ids, long delay);


#define ARRLEN(A) (sizeof(block_counts)/sizeof(block_counts[0]))

static void test_block_dist()
{
  static const int block_counts[] {30, 60, 120, 1024};
  const int max_block_count = block_counts[ARRLEN(block_counts) - 1];
  // static const int block_counts[] {4096};
  // static const int block_counts[] {3, 10, 100};
  // const std::array<int,3> block_counts {1, 10, 100, 1000};
  const long delay = 10*1000; // 10 us

  const int nsmid = query_nsmid();
  std::cout << "  %nsmid = " << nsmid << "\n";

  umem<ticks_t> tbuf(max_block_count);
  umem<uint32_t> idbuf(max_block_count);

  for (int block_count : block_counts) {
    std::cout << "============================================================\n";
    std::cout << block_count << " blocks\n";
    glob_block_dist<<<block_count,1>>>(tbuf, idbuf, delay); // warmup
    glob_block_dist<<<block_count,1>>>(tbuf, idbuf, delay);
    auto e = hipDeviceSynchronize();
    if (e != hipSuccess) {
      fatal(hipGetErrorName(e), " (" ,hipGetErrorString(e), "): unexpected error");
    }
    const rec r(block_count, idbuf, tbuf);

    r.format_block_distribution(std::cout);
    std::cout << "\n";

    std::ofstream of("events.json");
    r.format_block_distribution_json(of);
    // idbuf.str(std::cout, 8);
    // tbuf.str(std::cout, 4);

    std::cout << "\n";
  }
}

std::ostream &operator<<(std::ostream &os, const tstamps &ts) {
  os <<
    "{"
    "smid:" << ts.smid << ","
    "block_idx:" << ts.block_idx << ","
    "ticks:" << ts.ticks << ","
    "globaltimer:" << ts.globaltimer <<
    "}";
  return os;
}

//////////////////////////////////////////////////////////////////
// are timers persistant over dispatches?
// TEST: fetch time stamps, delay K, re-fetch; diff by SM
extern "C" __global__ void glob_get_times(tstamps *tss);


static void test_persist_dispatch()
{
  const int WARP_COUNT = 30;

  auto dispatch = [&](umem<tstamps> &tss) {
    glob_get_times<<<WARP_COUNT,1>>>(tss);
    auto e = hipDeviceSynchronize();
    if (e != hipSuccess) {
      fatal(hipGetErrorName(e), " (", hipGetErrorString(e), "): unexpected error");
    }
  };

  const int DISPATCHES = 10;
  std::vector<std::vector<tstamps>> tss;
  tss.reserve(DISPATCHES);
  // tss.emplace_back(WARP_COUNT);
  for (int i = 0; i < DISPATCHES; i++) {
    umem<tstamps> ts_buf(WARP_COUNT);
    if (i != 0) {
      // std::this_thread::sleep_for(2s);
      const int S = 1; // 60;
      std::cout << "waiting " << S << "s...";
      std::cout.flush();
      std::this_thread::sleep_for(std::chrono::seconds(S));
      std::cout << "\n";
    }
    std::cout << "dispatch" << i << "\n";

    dispatch(ts_buf);

    tss.emplace_back((std::vector<tstamps>)ts_buf);
  }

  auto findStamp = [&](smid_t smid, const std::vector<tstamps> &tss) -> const tstamps * {
    for (const auto &ts : tss) {
      if (ts.smid == smid) {
        return &ts;
      }
    }
    return nullptr;
  };

  const int KEY_COLS = 32;
  const int VAL_COLS = 12;
  auto emitStampTicks = [&](smid_t smid, bool globaltimer = true)
  {
    std::cout << std::setw(KEY_COLS) <<
      ("%smid=" + std::to_string(smid) + "/" + (globaltimer ? "%globaltimer" : "clock64()"));


    auto fmtDelta = [&](const tstamps *ts0, const tstamps *ts1) {
      std::cout << "  ";
      if (ts0 == nullptr || ts1 == nullptr) {
        std::cout << std::setw(VAL_COLS) << "???";
      } else {
        if (globaltimer) {
          std::cout << std::setw(VAL_COLS) << (ts1->globaltimer - ts0->globaltimer);
        } else {
          std::cout << std::setw(VAL_COLS) << (ts1->ticks - ts0->ticks);
        }
      }
    };

    for (int i = 1; i < DISPATCHES; i++) {
      const tstamps *ts0 = findStamp(smid, tss[i-1]);
      const tstamps *ts1 = findStamp(smid, tss[i-0]);
      fmtDelta(ts0, ts1);
    }
    std::cout << "\n";
  };

//  for (size_t i = 0; i < tss_buf0.size(); i++) {
//    std::cout << "==> " << tss_buf0[i] << "\n";
//  }

  auto fmtHeader = [&]() {
    std::cout << std::setw(KEY_COLS) << "deltas";
    for (int i = 0; i < DISPATCHES - 1; i++) {
      std::cout << "  " << std::setw(VAL_COLS) <<
        ("d" + std::to_string(i + 1) + "-d" + std::to_string(i));
    }
    std::cout << "\n";
  };

  // FIXME: assumes virtual SM ids are [0..WARP_COUNT-1] and persist to physical across calls
  std::cout << "============clock64()==========================================\n";
  fmtHeader();
  for (smid_t id = 0; id < WARP_COUNT; id++) {
    emitStampTicks(id, false);
  }
  std::cout << "\n";
  std::cout << "============%globaltimer=======================================\n";
  fmtHeader();
  for (smid_t id = 0; id < WARP_COUNT; id++) {
    emitStampTicks(id, true);
  }
}

static void test_persist_dispatch_proc()
{
  umem<tstamps> ts(1);
  glob_get_times<<<1,1>>>(ts);
  auto e = hipDeviceSynchronize();
  if (e != hipSuccess) {
    fatal(hipGetErrorName(e), " (", hipGetErrorString(e), "): unexpected error");
  }
  std::cout << "%globaltimer: " << std::setw(18) << ts[0].globaltimer << "\n";
}

static void test_delayed_clock64(int s)
{
  // stall, then init and call clock64
  // we should see about the same satartup time
  if (s > 0)
    std::this_thread::sleep_for(std::chrono::seconds(s));

  umem<tstamps> ts(1);
  glob_get_times<<<1,1>>>(ts);
  auto e = hipDeviceSynchronize();
  if (e != hipSuccess) {
    fatal(hipGetErrorName(e), " (", hipGetErrorString(e), "): unexpected error");
  }
  std::cout << "clock64(): " << std::setw(18) << ts[0].ticks << "\n";
}

static void test_print_clocks(int startup_delay)
{
  // stall, then init and call clock64
  // we should see about the same satartup time
  if (startup_delay > 0)
    std::this_thread::sleep_for(std::chrono::seconds(startup_delay));

  umem<tstamps> ts(1);
  glob_get_times<<<1,1>>>(ts);
  auto e = hipDeviceSynchronize();
  if (e != hipSuccess) {
    fatal(hipGetErrorName(e), " (", hipGetErrorString(e), "): unexpected error");
  }
  std::cout << "%globaltimer|%clock64:  " <<
    std::setw(18) << ts[0].globaltimer <<
    "  |  " <<
    std::setw(12) << ts[0].ticks <<
    "\n";
}

extern "C" __global__ void glob_globaltimer_cost(int *cost, int count, uint64_t *sum);
static void test_globaltimer_cost()
{
  const int WARP_COUNT = 30;
  const int LOOP_TRIPS = 32;

  umem<uint64_t> dummy(1);
  umem<int> deltas_buf(WARP_COUNT);

  glob_globaltimer_cost<<<WARP_COUNT,1>>>(deltas_buf, LOOP_TRIPS, dummy);

  auto st = statistics::construct<int>(deltas_buf);
  std::cout << "%globaltimer: takes " <<
    std::fixed << std::setprecision(3) << st.avg() <<
    " clocks on average\n";
}

// emit JSON
// https://www.gamasutra.com/view/news/176420/Indepth_Using_Chrometracing_to_view_your_inline_profiling_data.php
//
// https://docs.google.com/document/d/1CvAClvFfyA5R-PhYUmn5OOQtYMH4h6I0nSsKchNAySU/edit
//  ph=event type {B=begin,E=end,X=complete evetn,C=counter event,...,P=sample event}
//  ts=time, tts=thread clock timestamp
//  pid=process id
//  tid=thread id
//  args=[extra arguments]
//  {name:"warp start", }
// {"name": "myFunction", "cat": "foo", "ph": "X", "ts": 123, "dur": 234, "pid": 2343, "tid": 2347,
// "args": {
//   "first": 1
// }


/*
extern "C" __global__ void glob_block_dist_atomic(
  int64_t *times, uint32_t *ids, uint32_t *index, long delay);

static void test_block_dist_atomics()
{
}

extern "C" __global__ void glob_init(int64_t *times, uint32_t *ids, long delay);
extern "C" __global__ void glob_nanosleep(int64_t *times, uint32_t *ids, long delay);
extern "C" __global__ void glob_clock_value(int64_t *times, uint32_t *ids);

static void test_timer_latency()
{
  long delays[] = {0, 1000};
  const int block_counts[] {1, 10, 20};
  const int max_block_count = block_counts[ARRLEN(block_counts) - 1];

  std::vector<rec> records;

  umem<ticks_t> tbuf(max_block_count);
  umem<uint32_t> idbuf(max_block_count);
  for (long delay : delays) {
    // warmup
    for (int block_count : block_counts) {
  //    tbuf.prefetch_to_device();
  //    idbuf.prefetch_to_device();
      if (delay) {
        glob_nanosleep<<<block_count,1>>>(tbuf, idbuf, delay); // warmup
        glob_nanosleep<<<block_count,1>>>(tbuf, idbuf, delay);
      } else {
        glob_init<<<block_count,1>>>(tbuf, idbuf, delay); // warmup
        glob_init<<<block_count,1>>>(tbuf, idbuf, delay);
      }
      auto e = hipDeviceSynchronize();
      if (e != hipSuccess) {
        fatal(hipGetErrorName(e), " (" ,hipGetErrorString(e), "): unexpected error");
      }
      records.emplace_back(block_count, tbuf, idbuf);
      records.back().str(std::cout);
    }
  }
}
*/

int main(int argc, const char* argv[])
{
  // test_timer_latency(0, block_counts);
  if ((argc != 2 && argc != 3) || (argc == 1 &&
    (std::string(argv[1]) == "-h" || std::string(argv[1]) == "--help")))
  {
    std::cerr <<
      "usage: timers.exe TESTNAME\n"
      "where TESTNAME =\n"
      "    clocks [INT]\n"
      "  | delayed-clock64  INT\n"
      "  | dist\n"
      "  | globaltimer-cost\n"
      "  | persist\n"
      "  | print-timers\n"
      ;
    return EXIT_FAILURE;
  }
  int arg2 = 0;
  if (argc == 3) {
    size_t pos = 0;
    arg2 = std::stoi(argv[2], &pos, 10);
    if (pos == 0) {
      fatal(argv[2], "malformed integer argument");
    }
  }
  std::string test_name = argv[1];
  if (test_name == "clocks") {
    if (argc != 3) {
      arg2 = 0;
    }
    test_print_clocks(arg2);
  } else if (test_name == "delayed-clock64") {
    if (argc != 3) {
      std::cerr << "test requires INT argument (seconds)\n";
      return EXIT_FAILURE;
    }
    test_delayed_clock64(arg2);
  } else if (test_name == "dist") {
    test_block_dist();
  } else if (test_name == "globaltimer-cost") {
    test_globaltimer_cost();
  } else if (test_name == "persist") {
    test_persist_dispatch();
  } else if (test_name == "print-timers") {
    test_persist_dispatch_proc();
  } else {
    fatal(test_name, ": unsupported test");
  }

  return EXIT_SUCCESS;
}


// also