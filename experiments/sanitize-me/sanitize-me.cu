#include "hip/hip_runtime.h"
/// OPTIONS nvcc: -g -G
//
// Tests various compute sanitizer tools
// MemCheck
// https://docs.nvidia.com/compute-sanitizer/ComputeSanitizer/index.html#memcheck-tool
// TODO: test more memcheck tools
//    - mis-alignment
//    - uninit on device side (last elem and first elem)
#include "mincu.hpp"

#include <tuple>

using namespace mincu;


////////////////////////////////////////////////////////////////////////////////
struct opts {
  int verbosity = 0;

  bool normal() const {return verbosity >= 0;}
  bool verbose() const {return verbosity >= 1;}
  bool debug() const {return verbosity >= 2;}
  bool verbose_debug() const {return verbosity >= 3;}
};
constexpr opts DFT_OPTS;

////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void oob_glb_rd(uint32_t *oups, const uint32_t *inps)
{
  const size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  auto oob_val = inps[gid + 1]; // last one will be OOB
  oups[gid] = oob_val;
}
extern "C"
__global__ void oob_glb_wr(uint32_t *oups, const uint32_t *inps)
{
  const size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  auto val = inps[gid]; // last one will be OOB
  oups[gid + 1] = val;
}
extern "C"
__global__ void oob_shm_rd(uint32_t *oups, const uint32_t *inps)
{
  const size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ uint32_t smem[64];

  smem[threadIdx.x] = inps[gid];
  __syncthreads();

  auto oob_val = smem[threadIdx.x + 1]; // last one will be OOB
  oups[gid] = oob_val;
}
extern "C"
__global__ void oob_shm_wr(uint32_t *oups, const uint32_t *inps)
{
  const size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ uint32_t smem[64];

  smem[threadIdx.x - 1] = inps[gid];
  __syncthreads();

  auto oob_val = smem[threadIdx.x]; // last one will be OOB
  oups[gid] = oob_val;
}

extern "C"
__global__ void sqaure_kernel(uint32_t *oups, const uint32_t *inps)
{
  const size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  auto val = inps[gid];
  oups[gid] = val * val;
}

////////////////////////////////////////////////////////////////////////////////
using device_wrapper_t =
    std::function<void (size_t, size_t, uint32_t *, const uint32_t *)>;
using host_launcher_t = void (*)(const opts &, const char *, device_wrapper_t);
using test_t = std::tuple<const char *,host_launcher_t,device_wrapper_t>;

////////////////////////////////////////////////////////////////////////////////
static void launch_kernel(
    const opts &os, const char *tnm, device_wrapper_t wrapper)
{
  const size_t TPB = 64;
  const size_t BPG = 1;

  std::cout << "================ running " << tnm << "\n";

  const umem<uint32_t> inps {BPG * TPB, arith_seq<uint32_t>(0)};
  if (os.verbose_debug()) {
    std::cout << "INPS:\n";
    inps.str(std::cout, 8);
  }

  umem<uint32_t> oups {BPG * TPB};
  wrapper(BPG, TPB, oups, inps);
  auto e = hipDeviceSynchronize();
  if (e != hipSuccess) {
    fatal(hipGetErrorName(e), " (", hipGetErrorString(e), "): unexpected error");
  }
  if (os.debug()) {
    std::cout << "OUPS:\n";
    oups.str(std::cout, 8);
  }
}

////////////////////////////////////////////////////////////////////////////////
static void launch_kernel_uninit(
    const opts &os, const char *tnm, device_wrapper_t wrapper)
{
  const size_t TPB = 64;
  const size_t BPG = 1;

  std::cout << "================ running " << tnm << "\n";

  // const umem<uint32_t> inps {BPG * TPB, arith_seq<uint32_t>(0)};
  const umem<uint32_t> inps {BPG * TPB};
  if (os.verbose_debug()) {
    std::cout << "INPS:\n";
    inps.str(std::cout, 8);
  }

  umem<uint32_t> oups {BPG * TPB};
  wrapper(BPG, TPB, oups, inps);
  auto e = hipDeviceSynchronize();
  if (e != hipSuccess) {
    fatal(hipGetErrorName(e), " (", hipGetErrorString(e), "): unexpected error");
  }
  if (os.debug()) {
    std::cout << "OUPS:\n";
    oups.str(std::cout, 8);
  }
}

////////////////////////////////////////////////////////////////////////////////
static void launch_kernel_leak(
    const opts &os, const char *tnm, device_wrapper_t wrapper)
{
  const size_t TPB = 64;
  const size_t BPG = 1;

  std::cout << "================ running " << tnm << "\n";

  // const umem<uint32_t> inps {BPG * TPB, arith_seq<uint32_t>(0)};
  void *d_inps;
  CUDA_API(hipMalloc, &d_inps, BPG * TPB * sizeof(uint32_t));
  CUDA_API(hipMemset, d_inps, 42, BPG * TPB * sizeof(uint32_t));
  if (os.verbose_debug()) {
    std::cout << "INPS:\n" << "[[cannot show INPS on this test]]";
  }

  umem<uint32_t> oups {BPG * TPB};
  wrapper(BPG, TPB, oups, (const uint32_t *)d_inps);
  auto e = hipDeviceSynchronize();
  if (e != hipSuccess) {
    fatal(hipGetErrorName(e), " (", hipGetErrorString(e), "): unexpected error");
  }
  if (os.debug()) {
    std::cout << "OUPS:\n";
    oups.str(std::cout, 8);
  }
  // OOPS!
  // CUDA_API(hipFree, d_inps);
}


////////////////////////////////////////////////////////////////////////////////
static const test_t ALL_TESTS[] {
  {"oob-glb-rd", launch_kernel,
    [] (size_t bpg, size_t tpb, uint32_t *oups, const uint32_t *inps) {
      oob_glb_rd<<<bpg,tpb>>>(oups, inps);
    }},
  {"oob-glb-wr", launch_kernel,
    [] (size_t bpg, size_t tpb, uint32_t *oups, const uint32_t *inps) {
      oob_glb_wr<<<bpg,tpb>>>(oups, inps);
    }},
  {"oob-shm-rd", launch_kernel,
    [] (size_t bpg, size_t tpb, uint32_t *oups, const uint32_t *inps) {
      oob_shm_rd<<<bpg,tpb>>>(oups, inps);
    }},
  {"oob-shm-wr", launch_kernel,
    [] (size_t bpg, size_t tpb, uint32_t *oups, const uint32_t *inps) {
      oob_shm_wr<<<bpg,tpb>>>(oups, inps);
    }},
  {"uninit-glb", launch_kernel_uninit,
    [] (size_t bpg, size_t tpb, uint32_t *oups, const uint32_t *inps) {
      sqaure_kernel<<<bpg,tpb>>>(oups, inps);
    }},
  {"leak-glb", launch_kernel_leak,
    [] (size_t bpg, size_t tpb, uint32_t *oups, const uint32_t *inps) {
      sqaure_kernel<<<bpg,tpb>>>(oups, inps);
    }},
};


////////////////////////////////////////////////////////////////////////////////
int main(int argc, const char* argv[])
{
  std::string exe = argv[0]; // foo\\bar\\sanitize_me....exe
  auto last_slash = exe.find_last_of("/\\");
  if (last_slash != std::string::npos) {
    exe = exe.substr(last_slash + 1);
  }

  struct opts os = DFT_OPTS;
  std::vector<test_t> tests_to_run;

  for (int i = 1; i < argc; i++) {
    std::string arg = argv[i];
    std::string key = arg, val;
    auto eq = arg.find('=');
    if (eq != std::string::npos) {
      key = arg.substr(0, eq + 1); // include the =
      val = arg.substr(eq + 1);
    }
    auto bad_opt = [&](const char *msg) {
      fatal(arg, ": ", msg);
    };

    if (arg == "-h" || arg == "--help") {
      std::stringstream uss;
      uss <<
        "usage: " << exe << " [OPTS] TESTS\n"
        "where [OPTS]:\n"
        "  --list-tests          list all tests and exit 0\n"
        "  -v/-v2/-v3            verbosity/debug\n"
        "and TESTS are:\n"
        "          all       (runs all tests: will probably crash before running all)\n"
        "";
      for (auto [nm,_,__] : ALL_TESTS) {
        uss << "        | " << nm << "\n";
      }
      auto [first_nm,_,__] = ALL_TESTS[0];
      uss <<
        "EXAMPLES:\n"
        " % " << exe << " " << first_nm << "\n"
        "  runs this one test without sanitizer support\n"
        " % compute-sanitizer  --tool memcheck " << argv[0] << " " << first_nm << "\n"
        "  runs with sanitizer support\n"
        "";
      std::cout << uss.str();
      return EXIT_SUCCESS;
    } else if (arg == "--list-tests") {
      for (auto [tnm,_,__] : ALL_TESTS) {
        std::cout << tnm << "\n";
      }
      return EXIT_SUCCESS;
    } else if (arg == "-v0" || arg == "-v=0") {
      os.verbosity = 0;
    } else if (arg == "-v" || arg == "-v1" || arg == "-v=1") {
      os.verbosity = 1;
    } else if (arg == "-v2" || arg == "-v=2") {
      os.verbosity = 2;
    } else if (arg == "-v3" || arg == "-v=3") {
      os.verbosity = 3;
    } else if (arg == "all") {
      for (auto  t : ALL_TESTS) {
        tests_to_run.emplace_back(t);
      }
    } else if (arg.size() > 0 && arg[0] == '-') {
        bad_opt("invalid option");
    } else {
      bool found = false;
      for (auto [tnm,launcher,func] : ALL_TESTS) {
        if (tnm == arg) {
          tests_to_run.emplace_back(tnm, launcher, func);
          found = true;
          break;
        }
      }
      if (!found)
        fatal(arg, ": invalid test name (try -h)");
    }
  } // for

  if (tests_to_run.empty()) {
    fatal("expected test name (try -h)");
  }

  for (auto [tnm,launcher,func] : tests_to_run) {
    launcher(os, tnm, func);
  }

  return EXIT_SUCCESS;
}
