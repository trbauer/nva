#include "hip/hip_runtime.h"
#include "mincu.hpp"

#include <iostream>
#include <cstdio>
#include <functional>

using namespace mincu;

MINCU_ENABLE_COLOR_IO_VIA_STATIC_CONSTRUCTOR();

///////////////////////////////////////////
// TODO:
//  - move test files to testing.hpp (testing.cpp)
//  - move host-only tests to mincu-tests-host.cpp
//  - place device tests to mincu-tests-device.cpp
//  - mincu-tests.cu calls into both those
//     (all this breaks bexp.exe! unless we do #include only or something)
struct opts {
  int verbosity = 0;
};

static opts g_os;


template <typename T>
static std::string mc_format(T t, fmt_opts fos = fmt_opts())
{
  std::stringstream ss;
  format_elem<T>(ss, t, fos);
  return ss.str();
}

static std::string test_esc_str(std::string s) {
  std::stringstream ss;
  ss << '\"';
  for (size_t i = 0; i < s.size(); i++) {
    switch(s[i]) {
    case '\\': ss << "\\";
    case '\"': ss << "\"";
    case '\'': ss << "\'";
    case '\n': ss << "\n";
    case '\t': ss << "\t";
    default:
      if (std::isprint(s[i])) {
        ss << s[i];
      } else {
        ss << "\\x" << fmt_hex_digits(s[i], 2);
      }
    }
  }
  ss << '\"';
  return ss.str();
}
static std::string test_fmt(const char *str) {
  return test_esc_str(std::string(str));
}
static std::string test_fmt(std::string str) {
  return test_esc_str(str);
}

template <typename T>
static std::string test_fmt(const T &t) {
  return mincu::format(t);
}



static const char *g_current_label = "?";

static void test_fatal(
    int line, const char *macro,
    const char *sut_expr, const std::string &sut_fmtd,
    const char *exp_expr, const std::string &exp_fmtd,
    std::string hint = "")
{
  mincu::fatal("test ", ansi_red(g_current_label), "; near line ",
               ansi_yellow(line), ": ", macro, "(", sut_expr, ", ", exp_expr,
               ")\n"
               "    sut:",
               sut_fmtd, ", exp:", exp_fmtd, (hint.empty() ? "" : "\n  "), hint);
}

template <typename T1, typename T2>
static void test_eq_impl(
    int line,
    const char *sut_expr, const T1 &sut,
    const char *exp_expr, const T2 &exp, std::string hint = "")
{
  if (!(sut == exp)) {
    test_fatal(line, "TEST_EQ", sut_expr, test_fmt(sut), exp_expr,
               test_fmt(exp), hint);
  }
}
template <typename T1, typename T2>
static void test_ne_impl(
    int line,
    const char *sut_expr, const T1 &sut,
    const char *exp_expr, const T2 &exp)
{
  if (!(sut != exp)) {
    test_fatal(line, "TEST_NE", sut_expr, test_fmt(sut), exp_expr,
               test_fmt(exp));
  }
}
template <typename T1, typename T2>
static void test_lt_impl(
    int line,
    const char *sut_expr, const T1 &sut,
    const char *exp_expr, const T2 &exp)
{
  if (!(sut < exp)) {
    test_fatal(line, "TEST_NE", sut_expr, test_fmt(sut), exp_expr,
               test_fmt(exp));
  }
}
template <typename T1, typename T2>
static void test_le_impl(
    int line,
    const char *sut_expr, const T1 &sut,
    const char *exp_expr, const T2 &exp)
{
  if (!(sut <= exp)) {
    test_fatal(line, "TEST_NE", sut_expr, test_fmt(sut), exp_expr,
               test_fmt(exp));
  }
}
template <typename T1, typename T2>
static void test_gt_impl(
    int line,
    const char *sut_expr, const T1 &sut,
    const char *exp_expr, const T2 &exp)
{
  if (!(sut > exp)) {
    test_fatal(line, "TEST_GT", sut_expr, test_fmt(sut), exp_expr,
               test_fmt(exp));
  }
}
template <typename T1, typename T2>
static void test_ge_impl(
    int line,
    const char *sut_expr, const T1 &sut,
    const char *exp_expr, const T2 &exp)
{
  if (!(sut >= exp)) {
    test_fatal(line, "TEST_GE", sut_expr, test_fmt(sut), exp_expr,
               test_fmt(exp));
  }
}
#define TEST_EQ(SUT, EXP, ...) \
  test_eq_impl(__LINE__, #SUT, SUT, #EXP, EXP, __VA_ARGS__)
#define TEST_NE(SUT, EXP) \
  test_ne_impl(__LINE__, #SUT, SUT, #EXP, EXP)
#define TEST_LT(SUT, EXP) \
  test_lt_impl(__LINE__, #SUT, SUT, #EXP, EXP)
#define TEST_LE(SUT, EXP) \
  test_le_impl(__LINE__, #SUT, SUT, #EXP, EXP)
#define TEST_GT(SUT, EXP) \
  test_gt_impl(__LINE__, #SUT, SUT, #EXP, EXP)
#define TEST_GE(SUT, EXP) \
  test_ge_impl(__LINE__, #SUT, SUT, #EXP, EXP)

static void start_test(const char *lbl)
{
  g_current_label = lbl;
  if (g_os.verbosity >= 0)
    std::cout << "============= " << lbl << "\n";
}

static void TEST_GROUP(const char *LBL, std::function<void()> BLOCK) {
  start_test(LBL);
  BLOCK();
  g_current_label = "?";
}


/*
extern "C"
__global__ void add_float_k(
    float *dsts,
    const float *srcs, float k)
{
  const int gid = blockIdx.x * blockDim.x + threadIdx.x;
  auto val = srcs[gid];
  auto val_plus_k = val + k;
  dsts[gid] = val_plus_k;
}


static void test_add_float_k()
{
  start_test("static void test_add_float_k");
  static const size_t BLOCKS = 1; // 1 warp only
  static const size_t TPB = 32; // threads per block (1 warp)

  umem<float> inps(64, arith_seq<float>(0.0f));
  umem<float> oups(64);
  inps.str(std::cout, 8, 3);

  add_float_k<<<BLOCKS,TPB>>>(dsts, srcs, 1.0f);
  auto e = hipDeviceSynchronize();
  if (e != hipSuccess) {
    mincu::fatal(hipGetErrorName(e), " (", hipGetErrorString(e), "): unexpected error");
  }
  oups.str(std::cout, 8, 3);
} // test_add_float_k
*/

static void run_buf_init_tests()
{
  /////////////////////////////////////////////
  // constant sequence initializer
  TEST_GROUP("umem<unsigned>.no_init",
      [] {
        umem<unsigned> buf(8);
        TEST_EQ(buf.size(), 8);
      });

  TEST_GROUP("umem<unsigned>.const_seq(3).reinit(const_seq(7)",
      [] {
        umem<unsigned> buf(8, const_seq(3u));
        for (size_t i = 0; i < buf.size(); i++) {
          TEST_EQ(buf[i], 3u);
        }
        // re-initialize
        buf.init(const_seq(7u));
        for (size_t i = 0; i < buf.size(); i++) {
          TEST_EQ(buf[i], 7u);
        }
      });

  TEST_GROUP("umem<unsigned>.const_seq(3)[2] = 55",
      [] {
        umem<unsigned> buf(8, const_seq(3u));
        buf[2] = 55;
        for (size_t i = 0; i < buf.size(); i++) {
          if (i == 2) {
            TEST_EQ(buf[i], 55u);
          } else {
            TEST_EQ(buf[i], 3u);
          }
        }
      });

  TEST_GROUP("umem<uint2>.const_seq(2, 3)[3] = make_uint2(4, 1)",
      [] {
        umem<uint2> buf(4, const_seq(make_uint2(2, 3)));
        buf[3] = make_uint2(4, 1);
        for (size_t i = 0; i < buf.size(); i++) {
          if (i == 3) {
            TEST_EQ(buf[i], make_uint2(4, 1));
          } else {
            TEST_EQ(buf[i], make_uint2(2, 3));
          }
        }
      });

  TEST_GROUP("umem<unsigned>.arit_seq(0)",
      [] {
        // arithmetic sequence initializer
        umem<unsigned> buf {8, arith_seq(0u)};
        for (size_t i = 0; i < buf.size(); i++) {
          TEST_EQ(buf[i], i);
        }
      });
  TEST_GROUP("umem<unsigned>.arit_seq(0, 2)",
      [] {
        // arithmetic sequence initializer
        umem<unsigned> buf {8, arith_seq(0u, 2u)};
        for (size_t i = 0; i < buf.size(); i++) {
          TEST_EQ(buf[i], 2 * i);
        }
      });
  TEST_GROUP("umem<unsigned>.arit_seq(77u, 2u, 5u)",
      [] {
        umem<unsigned> buf {8, arith_seq(77u, 2u, 5u)};
        unsigned val = 77u % 5;
        for (size_t i = 0; i < buf.size(); i++) {
          TEST_EQ(buf[i], val);
          val = (val + 2u) % 5u;
        }
      });

  TEST_GROUP("umem<uint2>.arith_seq(make_uint2(0, 1), make_uint2(1, 2))",
      [] {
        auto exp = make_uint2(0, 1);
        const auto delta = make_uint2(1, 2);
        umem<uint2> buf {8, arith_seq(exp, delta)};
        for (size_t i = 0; i < buf.size(); i++) {
          TEST_EQ(buf[i], exp);
          exp += delta;
        }
      });
  TEST_GROUP("umem<uint2>.arith_seq(make_uint2(4,2), make_uint2(1,2), make_uint2(3,4))",
      [] {
        auto exp = make_uint2(4, 2);
        const auto delta = make_uint2(1, 2), mod = make_uint2(3, 4);
        umem<uint2> buf {8, arith_seq(exp, delta, mod)};
        exp = exp % mod;
        TEST_EQ(buf[0], make_uint2(1, 2)); // inital value is mod'd
        for (size_t i = 0; i < buf.size(); i++) {
          TEST_EQ(buf[i], exp, format("idx[", i ,"]"));
          exp += delta;
          exp %= mod;
        }
      });

    // this assumes std::mt19937 is deterministic across STL implementations
    // (I *think* this is true.  I think the seeds are give in the spec.)
    TEST_GROUP("umem<unsigned>.rnd_seq(0, 10)",
      [] {
        random_state rs {12007};
        umem<unsigned> buf {4u, rnd_seq(rs, 0u, 10u)};
        // buf.str(std::cout);
        TEST_EQ(buf[0], 1);
        TEST_EQ(buf[1], 7);
        TEST_EQ(buf[2], 8);
        TEST_EQ(buf[3], 1);
      });

  ///////////////////////////////////////////////
  // TODO: extra tests
  // - init with lambda
  // - init cyc
  // - init with random (fix values), ensure random doesn't get clobbered
  // - init with random int8_t
  // - init float types (fmod in arith_seq)
  //
  // - umem::str() (format_elem etc..)
  //
} // test_inits


static void run_format_tests()
{
  TEST_GROUP("format.misc",
      [] {
        TEST_EQ(format((const char *)nullptr), "nullptr"); // doesn't blow how
        TEST_EQ(format(2), "2");
        TEST_EQ(format(2u), "0x2");
        TEST_EQ(format(-2," ",2u), "-2 0x2");
        TEST_EQ(format(hex(2)), "00000002"); // hex() wrapper does not include 0x
        TEST_EQ(format(hex(2, 4)), "0002");
      });

  /////////////////////////////////////////////////////////////////////////////
  // char*
  TEST_GROUP("mctypes.format_elem<int8_t>",
      [] {
        const fmt_opts fos {};
        TEST_EQ(mc_format<int8_t>(0), "0");
        TEST_EQ(mc_format<int8_t>(0, fos.hex()), "0x00");
        TEST_EQ(mc_format<int8_t>(31), "31");
        TEST_EQ(mc_format<int8_t>(-16), "-16");
        TEST_EQ(mc_format<int8_t>(127), "127");
        TEST_EQ(mc_format<int8_t>(-128), "-128");
        TEST_EQ(mc_format<int8_t>(0x10), "16");
        TEST_EQ(mc_format<int8_t>(16, fos.hex()), "0x10");
        TEST_EQ(mc_format<int8_t>(16, fos.cols(0)), "16");
        TEST_EQ(mc_format<int8_t>(16, fos.cols(1)), "16");
        TEST_EQ(mc_format<int8_t>(16, fos.cols(-2)), "16");
        TEST_EQ(mc_format<int8_t>(16, fos.cols(4)), "  16");
        TEST_EQ(mc_format<int8_t>(18, fos.cols(5).hex()), " 0x12");
        TEST_EQ(mc_format<int8_t>(-4, fos.cols(5).hex()), " 0xFC");
        TEST_EQ(mc_format<int8_t>(-128, fos.hex()), "0x80");
        TEST_EQ(mc_format<int8_t>(-127, fos.hex()), "0x81");
        TEST_EQ(mc_format<int8_t>(127, fos.hex()), "0x7F");
      });
  TEST_GROUP("mctypes.format_elem<char2>",
      [] {
        const fmt_opts fos {};
        TEST_EQ(mc_format<char2>(make_char2(1,22)), "{1,22}");
        TEST_EQ(mc_format<char2>(make_char2(1,22), fos.cols(4)), "{   1,  22}");
      });
  TEST_GROUP("mctypes.format_elem<char3>",
      [] {
        const fmt_opts fos {};
        TEST_EQ(mc_format<char3>(make_char3(1,22,3)), "{1,22,3}");
        TEST_EQ(mc_format<char3>(make_char3(1,22,3), fos.cols(2)), "{ 1,22, 3}");
      });
  TEST_GROUP("mctypes.format_elem<char4>",
      [] {
        const fmt_opts fos {};
        TEST_EQ(mc_format<char4>(make_char4(1,22,3,44)), "{1,22,3,44}");
        TEST_EQ(mc_format<char4>(make_char4(1,22,3,124), fos.cols(3)), "{  1, 22,  3,124}");
        TEST_EQ(mc_format<char4>(make_char4(-128,2,18,3), fos.hex().cols(5)), "{ 0x80, 0x02, 0x12, 0x03}");
      });

  /////////////////////////////////////////////////////////////////////////////
  // uchar*
  TEST_GROUP("mctypes.format_elem<uint8_t>",
      [] {
        const fmt_opts fos {};
        TEST_EQ(mc_format<uint8_t>(0x0), "0x00");
        TEST_EQ(mc_format<uint8_t>(0x3, fos.cols(0)), "0x03");
        TEST_EQ(mc_format<uint8_t>(0x3, fos.dec()), "3");
        TEST_EQ(mc_format<uint8_t>(0x11, fos.dec()), "17");
        TEST_EQ(mc_format<uint8_t>(0x11, fos.dec().cols(5)), "   17");
        TEST_EQ(mc_format<uint8_t>(0x34), "0x34");
        TEST_EQ(mc_format<uint8_t>(0xFF), "0xFF");
        TEST_EQ(mc_format<uint8_t>(0xFF, fos.dec()), "255");
        TEST_EQ(mc_format<uint8_t>(0x34, fos.cols(6)), "  0x34");
      });
  TEST_GROUP("mctypes.format_elem<uchar2>",
      [] {
        TEST_EQ(mc_format<uchar2>(make_uchar2(1,2)), "{0x01,0x02}");
      });
  TEST_GROUP("mctypes.format_elem<uchar3>",
      [] {
        TEST_EQ(mc_format<uchar3>(make_uchar3(1,2,3)), "{0x01,0x02,0x03}");
      });
  TEST_GROUP("mctypes.format_elem<uchar4>",
      [] {
        TEST_EQ(mc_format<uchar4>(make_uchar4(1,2,3,4)), "{0x01,0x02,0x03,0x04}");
      });
}

static void run_mc_derived_function_tests()
{
  TEST_GROUP("mctypes.derived<uint2>",
      [] {
        TEST_EQ(make_uint2(1, 2), make_uint2(1, 2));
        TEST_NE(make_uint2(1, 2), make_uint2(1, 3));
        TEST_NE(make_uint2(1, 2), make_uint2(2, 2));
        //
        TEST_EQ(make_uint2(2, 2), 2u);
        TEST_NE(make_uint2(2, 3), 2u);
        //
        TEST_EQ(make_uint2(2, 2) + make_uint2(3, 4), make_uint2(5, 6));
        TEST_EQ(make_uint2(2, 2) + 1u,               make_uint2(3, 3));
        TEST_EQ(2u * make_uint2(2, 3),               make_uint2(4, 6));
        TEST_EQ(make_int2(2, 3) * -2,                make_int2(-4, -6));
        TEST_EQ(make_uint2(6, 7) % 2u,               make_uint2(0, 1));

        // TODO: many many many more (all the dervied operators etc..)
      });
}

int main(int argc, char **argv)
{
  for (int i = 1; i < argc; i++) {
    std::string arg = argv[i];
    std::string key = arg, val;
    auto eq = arg.find('=');
    if (eq != std::string::npos) {
      key = arg.substr(0, eq + 1); // include the =
      val = arg.substr(eq + 1);
    }
    auto bad_opt = [&](std::string msg0, std::string msg1 = "", std::string msg2 = "") {
      fatal(arg, ": ", msg0, msg1, msg2);
    };

    if (arg == "-h" || arg == "--help") {
      std::cout <<
        "usage: mincu-tests.exe OPTS TEST+\n"
        "where OPTS are:\n"
        "  -v/-v2                  sets verbosity\n"
        "EXAMPLES:\n"
        " % ...\n"
        "";
      return EXIT_SUCCESS;
    } else if (arg == "-v") {
      g_os.verbosity = 1;
    } else if (arg == "-v2") {
      g_os.verbosity = 2;
    } else if (!arg.empty() && arg[0] == '-') {
      bad_opt("unexpected option");
    } else {
      bad_opt("unexpected argument");
    }
  } // for args

  run_format_tests();

  run_mc_derived_function_tests();

  run_buf_init_tests();
  // TODO: run_format_umem_tests()
  //  test_add_float_k();

  return EXIT_SUCCESS;
}

/*

1>
1>E:\dev\nvaa\experiments\mincu-tests\vs>"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.3\bin\nvcc.exe" -gencode=arch=compute_75,code=\"compute_75,compute_75\" -gencode=arch=compute_75,code=\"sm_75,compute_75\" --use-local-env -ccbin "C:\Program Files\Microsoft Visual Studio\2022\Professional\VC\Tools\MSVC\14.38.33130\bin\HostX64\x64" -x cu -rdc=true  -I"E:\dev\nvaa\experiments\mincu-tests\.." -I"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.3\include"     --keep-dir x64\Debug  -maxrregcount=0   --machine 64 --compile -cudart static -std=c++17 -Xcompiler="/EHsc -Zi -Ob0" -g  -D_WINDOWS -D"CMAKE_INTDIR=\"Debug\"" -D_MBCS -D"CMAKE_INTDIR=\"Debug\"" -Xcompiler "/EHsc /W1 /nologo /Od /FS /Zi /RTC1 /MDd " -Xcompiler "/Fdmincu-tests75.dir\Debug\vc143.pdb" -o mincu-tests75.dir\Debug\mincu-tests.obj "E:\dev\nvaa\experiments\mincu-tests\mincu-tests.cu"
1>mincu-tests.cu
1>tmpxft_00004328_00000000-7_mincu-tests.cudafe1.cpp
1>
1>E:\dev\nvaa\experiments\mincu-tests\vs>"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.3\bin\nvcc.exe"\
  -dlink  -o mincu-tests75.dir\Debug\mincu-tests75.device-link.obj -Xcompiler "/EHsc /W1 /nologo /Od /Zi /RTC1 /MDd \
  " -Xcompiler "/Fdmincu-tests75.dir\Debug\vc143.pdb" -L"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.3\bin/crt" \
  -L"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.3\lib\x64" \
  cudadevrt.lib cudart_static.lib kernel32.lib user32.lib gdi32.lib winspool.lib shell32.lib ole32.lib \
    oleaut32.lib uuid.lib comdlg32.lib advapi32.lib \
  -forward-unknown-to-host-compiler -Wno-deprecated-gpu-targets \
  -gencode=arch=compute_75,code=compute_75 -gencode=arch=compute_75,code=sm_75
  "mincu-tests75.dir\Debug\mincu-tests.obj"

1>E:\dev\nvaa\experiments\mincu-tests\vs>"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.3\bin\nvcc.exe" \
  --use-local-env \
  -ccbin "C:\Program Files\Microsoft Visual Studio\2022\Professional\VC\Tools\MSVC\14.38.33130\bin\HostX64\x64" \
  -x cu -rdc=true  -I"E:\dev\nvaa\experiments\mincu-tests\.." -I"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.3\include" \
  --keep-dir x64\Debug  -maxrregcount=0   --machine 64 --compile -cudart static -std=c++20 \
  --generate-code=arch=compute_75,code=[compute_75,sm_75] -Xcompiler="/EHsc -Ob0 \
  -Zi" -g  -D_WINDOWS -D"CMAKE_INTDIR=\"Debug\"" -D_MBCS -D"CMAKE_INTDIR=\"Debug\"" \
  -Xcompiler "/EHsc /W1 /nologo /Od /FS /Zi /RTC1 /MDd " -Xcompiler "/Fdmincu-tests75.dir\Debug\vc143.pdb" \
  -o mincu-tests75.dir\Debug\mincu-tests.obj \
  "E:\dev\nvaa\experiments\mincu-tests\mincu-tests.cu"

*/

/*
// FAILS to repro with custom type here
// uint2 still failed on format, but not xformat here
// It seems that the definition of (x)format_to must follow the << instances.
template <typename...Ts>
static inline void xformat_to(std::ostream &os) { }
template <typename T, typename...Ts>
static inline void xformat_to(std::ostream &os, T t, Ts...ts) {os << t; format_to(os, ts...);}
template <typename...Ts>
static inline std::string xformat(Ts...ts) {
  std::stringstream ss; xformat_to(ss, ts...); return ss.str();
}
struct foo {
  int value;
  foo(int v) : value(v) { }
};

static inline std::ostream &operator <<(std::ostream &os, foo f) {
  os << "foo{" << f.value << "}";
  return os;
}

static void micro_test()
{
  std::cout << foo(22) << "\n";
  std::cout << xformat("bar ", 32, " baz") << "\n";
  std::cout << xformat("qux ", foo(32), " zap") << "\n";
  uint2 i2 = make_uint2(1, 2);
  std::cout << "uint2 " << i2 << "\n";
  std::cout << "uint2 " << xformat(i2) << "\n";
  std::cout << "uint2 " << mincu::format(i2) << "\n";
}
// */

