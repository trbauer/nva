#include "hip/hip_runtime.h"
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#using-tma-to-transfer-multi-dimensional-arrays

#include <hip/hip_runtime.h>         // CUtensormap
#include <cuda/barrier>
using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

static int const SMEM_WIDTH = 64;
static int const SMEM_HEIGHT = 64;

__global__ void kernel(
    const __grid_constant__ CUtensorMap tensor_map, int x, int y)
{
  // The destination shared memory buffer of a bulk tensor operation should be
  // 128 byte aligned.
  __shared__ alignas(128) int smem_buffer[SMEM_HEIGHT][SMEM_WIDTH];

  // Initialize shared memory barrier with the number of threads participating in the barrier.
  #pragma nv_diag_suppress static_var_with_dynamic_init
  __shared__ barrier bar;

  if (threadIdx.x == 0) {
    // Initialize barrier. All `blockDim.x` threads in block participate.
    init(&bar, blockDim.x);
    // Make initialized barrier visible in async proxy.
    cde::fence_proxy_async_shared_cta();
  }
  // Syncthreads so initialized barrier is visible to all threads.
  __syncthreads();

  barrier::arrival_token token;
  if (threadIdx.x == 0) {
    // Initiate bulk tensor copy.
    cde::cp_async_bulk_tensor_2d_global_to_shared(&smem_buffer, &tensor_map, x, y, bar);
    // Arrive on the barrier and tell how many bytes are expected to come in.
    token = cuda::device::barrier_arrive_tx(bar, 1, sizeof(smem_buffer));
  } else {
    // Other threads just arrive.
    token = bar.arrive();
  }
  // Wait for the data to have arrived.
  bar.wait(std::move(token));

  // Symbolically modify a value in shared memory.
  smem_buffer[0][threadIdx.x] += threadIdx.x;

  // Wait for shared memory writes to be visible to TMA engine.
  cde::fence_proxy_async_shared_cta();
  __syncthreads();
  // After syncthreads, writes by all threads are visible to TMA engine.

  // Initiate TMA transfer to copy shared memory to global memory
  if (threadIdx.x == 0) {
    cde::cp_async_bulk_tensor_2d_shared_to_global(&tensor_map, x, y, &smem_buffer);
    // Wait for TMA transfer to have finished reading shared memory.
    // Create a "bulk async-group" out of the previous bulk copy operation.
    cde::cp_async_bulk_commit_group();
    // Wait for the group to have completed reading from shared memory.
    cde::cp_async_bulk_wait_group_read<0>();
  }

  // Destroy barrier. This invalidates the memory region of the barrier. If
  // further computations were to take place in the kernel, this allows the
  // memory location of the shared memory barrier to be reused.
  if (threadIdx.x == 0) {
    (&bar)->~barrier();
  }
}

/*
int main()
{
  CUtensorMap tensor_map{};
  // rank is the number of dimensions of the array.
  constexpr uint32_t rank = 2;
  uint64_t size[rank] = {GMEM_WIDTH, GMEM_HEIGHT};
  // The stride is the number of bytes to traverse from the first element of one row to the next.
  // It must be a multiple of 16.
  uint64_t stride[rank - 1] = {GMEM_WIDTH * sizeof(int)};
  // The box_size is the size of the shared memory buffer that is used as the
  // destination of a TMA transfer.
  uint32_t box_size[rank] = {SMEM_WIDTH, SMEM_HEIGHT};
  // The distance between elements in units of sizeof(element). A stride of 2
  // can be used to load only the real component of a complex-valued tensor, for instance.
  uint32_t elem_stride[rank] = {1, 1};

  // Get a function pointer to the cuTensorMapEncodeTiled driver API.
  auto cuTensorMapEncodeTiled = get_cuTensorMapEncodeTiled();

  // Create the tensor descriptor.
  hipError_t res = cuTensorMapEncodeTiled(
    &tensor_map,                // CUtensorMap *tensorMap,
    CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_INT32,
    rank,                       // cuuint32_t tensorRank,
    tensor_ptr,                 // void *globalAddress,
    size,                       // const cuuint64_t *globalDim,
    stride,                     // const cuuint64_t *globalStrides,
    box_size,                   // const cuuint32_t *boxDim,
    elem_stride,                // const cuuint32_t *elementStrides,
    // Interleave patterns can be used to accelerate loading of values that
    // are less than 4 bytes long.
    CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
    // Swizzling can be used to avoid shared memory bank conflicts.
    CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
    // L2 Promotion can be used to widen the effect of a cache-policy to a wider
    // set of L2 cache lines.
    CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
    // Any element that is outside of bounds will be set to zero by the TMA transfer.
    CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
  );

  kernel<<<1,1>>>(tensor_map);
}
*/