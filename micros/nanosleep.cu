
#include <hip/hip_runtime.h>

// D.run ["micros/nanosleep.cu","--arch=sm_75","-o=micros/sm_75/nanosleep.sass","-lines"]

extern "C" __global__ void nanosleep_kernel(
    volatile float *A,
    float *OUT,
    int k)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  float sum = A[id];
  __nanosleep(0x12345678);
  for (int i = 0; i < k; i++) {
    // asm volatile("nanosleep.u32 %0;" :: "r"(ns));
    sum += A[id];
    __nanosleep(i);
  }
  OUT[id] = sum;
}